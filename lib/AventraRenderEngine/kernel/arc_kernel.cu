#include <../include/glad/glad.h>
#include <cuda_gl_interop.h>
#include <iostream>
#include <../include/Aventra_arc.hpp>
#define GL_GLEXT_PROTOTYPES


using namespace Aventra;

const char *vertexShaderSource2 = "#version 330 core\n"
    "layout (location = 0) in vec3 aPos;\n"
    "void main()\n"
    "{\n"
    "   gl_Position = vec4(aPos.x, aPos.y, aPos.z, 1.0);\n"
    "}\0";

const char *fragmentShaderSource2 = "#version 330 core\n"
    "out vec4 FragColor;\n"
    "void main()\n"
    "{\n"
    "   FragColor = vec4(1.0f, 1.0f, 0.2f, 1.0f);\n"
    "}\n\0";


inline unsigned int compileShaders() {
  unsigned int vertexShader = glCreateShader(GL_VERTEX_SHADER);
  glShaderSource(vertexShader, 1, &vertexShaderSource2, NULL);
  glCompileShader(vertexShader);

  int success;
  char infoLog[512];
  glGetShaderiv(vertexShader, GL_COMPILE_STATUS, &success);
  if (!success) {
    glGetShaderInfoLog(vertexShader, 512, NULL, infoLog);
    std::cout << "ERROR::SHADER::VERTEX::COMPILATION_FAILED\n" << infoLog << std::endl;
    //WDS::Error("Failed to compile Vertex shader", 30, false).Log();
   }

  unsigned int fragmentShader = glCreateShader(GL_FRAGMENT_SHADER);
  glShaderSource(fragmentShader, 1, &fragmentShaderSource2, NULL);
  glCompileShader(fragmentShader);
  glGetShaderiv(fragmentShader, GL_COMPILE_STATUS, &success);
  if (!success) {
    glGetShaderInfoLog(fragmentShader, 512, NULL, infoLog);
    std::cout << "ERROR::SHADER::FRAGMENT::COMPILATION_FAILED\n" << infoLog << std::endl;
	//WDS::Error("Failed to compile Fragment Shader", 31, false).Log();
  }

      // link the shaders
  unsigned int shaderProgram = glCreateProgram();
  glAttachShader(shaderProgram, vertexShader);
  glAttachShader(shaderProgram, fragmentShader);
  glLinkProgram(shaderProgram);
  glGetProgramiv(shaderProgram, GL_LINK_STATUS, &success);

  if (!success) {
    glGetProgramInfoLog(shaderProgram, 512, NULL, infoLog);
    std::cout << "ERROR::SHADER::PROGRAM::LINKING_FAILED\n" << infoLog << std::endl;
  } else {
    //std::cout << "Linked Shaders Sucessfully\n";
  }

  //clean up shaders
  glDeleteShader(vertexShader);
  glDeleteShader(fragmentShader);

  return shaderProgram;
}



Arc::Arc(float r, float cx, float cy) {

  this->shaderProgram = compileShaders();
  this->r = r;
  this->cx = cx;
  this->cy = cy;

  // Allocate the VBO
  glGenBuffers(1, &this->VBO);
  glBindBuffer(GL_ARRAY_BUFFER, this->VBO);
  unsigned int size = 90 * 4 * sizeof(float); // 90, 4 floats
  glBufferData(GL_ARRAY_BUFFER, size, NULL, GL_DYNAMIC_DRAW);
  glBindBuffer(GL_ARRAY_BUFFER, 0);

  hipGraphicsResource_t VBO_Cuda;
  
  // register with CUDA
  hipGraphicsGLRegisterBuffer(&VBO_Cuda, this->VBO, cudaGraphicsMapFlagsWriteDiscard);
  hipGraphicsMapResources(1, &VBO_Cuda, 0);
  size_t numBytes;
  hipGraphicsResourceGetMappedPointer((void **)&this->VBO, &numBytes, VBO_Cuda);
  
}
